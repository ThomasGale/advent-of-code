#include "hip/hip_runtime.h"
#pragma once

#include "Common.hpp"
#include "cudautils.cuh"

namespace aoc {
namespace y2020 {

namespace d01impl {
__global__ void sumMatchMul(int n, uint* in, int target, uint* ans) {
    using namespace cudautils::semaphore;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        for (int j = i; j < n; ++j) {
            for (int k = j; k < n; ++k) {
                if (in[i] + in[j] + in[k] == target) {
                    __syncthreads();
                    if (threadIdx.x == 0)
                        acquire_semaphore(&sem);
                    __syncthreads();

                    // Single Thread here
                    *ans = in[i] * in[j] * in[k];

                    __threadfence();
                    __syncthreads();
                    if (threadIdx.x == 0)
                        release_semaphore(&sem);
                    __syncthreads();
                }
            }
        }
    }
}
} // namespace d01impl

class d01 : public Solution {
  public:
    static std::unique_ptr<Solution> create() {
        return std::make_unique<d01>();
    }

    void Calculate(std::istream& input) override {
        using namespace d01impl;
        int target = 2020;
        std::vector<std::string> inputStrs = utils::reader::read_input(input);

        // Init data.
        uint n = inputStrs.size();
        uint* in;
        hipMallocManaged(&in, n * sizeof(uint));

        for (int i = 0; i < n; ++i) {
            in[i] = uint(std::stoi(inputStrs[i]));
        }

        // Get cuda properies for the current device.
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        uint blockSize = prop.maxThreadsPerBlock;
        uint numBlocks = (n + blockSize - 1) / blockSize;

        // Run
        uint* result;
        hipMallocManaged(&result, sizeof(uint));
        *result = 0;
        sumMatchMul<<<numBlocks, blockSize>>>(n, in, target, result);

        // Wait for GPU to finish before accessing on host
        hipDeviceSynchronize();

        // Part 1
        std::cout << *result << std::endl;

        hipFree(in);
        hipFree(result);
    }
};

} // namespace y2020
} // namespace aoc

#include "hip/hip_runtime.h"
#pragma once

#include "Common.hpp"

namespace aoc {
namespace y2020 {



__global__ void sumMatchMul(int n, uint* in, int target, uint* ans) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        for (int j = i; j < n; ++j) {
            if (in[i] + in[j] == target) {
                *ans = in[i] * in[j];
            }
        }
    }
}

class d01 : public Solution {
  public:
    static std::unique_ptr<Solution> create() {
        return std::make_unique<d01>();
    }

    void Calculate(std::istream& input) override {
        int target = 2020;
        std::vector<std::string> inputStrs = utils::reader::read_input(input);

        // Init data.
        uint n = inputStrs.size();
        uint* in;
        hipMallocManaged(&in, n * sizeof(uint));

        for (int i = 0; i < n; ++i) {
            in[i] = uint(std::stoi(inputStrs[i]));
        }

        // Get cuda properies for the current device.
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        uint blockSize = prop.maxThreadsPerBlock;
        uint numBlocks = (n + blockSize - 1) / blockSize;

        // Run
        uint* result;
        hipMallocManaged(&result, sizeof(uint));
        sumMatchMul<<<numBlocks, blockSize>>>(n, in, target, result);

        // Wait for GPU to finish before accessing on host
        hipDeviceSynchronize();

        // Part 1
        std::cout << *result << std::endl;

        hipFree(in);
    }
};

} // namespace y2020
} // namespace aoc
